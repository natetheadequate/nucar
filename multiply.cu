#include "hip/hip_runtime.h"
#include <cstdio>

#include "dotproductmatrices.c"
// should define matrices A and B, where A is an array of N arrays of length M, and B is an array of M arrays of length P

__global__ void multiplyVec(int* A, int* B, int* result){
  printf("oh");
  for(int i=0; i<threadIdx.x*20;i++){
    printf("%i", *A);
  }
  result[threadIdx.x] = A[threadIdx.x] * B[threadIdx.x];
  printf("hi");
  return;
}

__global__ void sayHi(int* A, int* B, int* result){
  printf("yo");
  return;
}

__global__ void dotproduct(int* A, int* B, int N, int* result){
  int* C;
  printf("%i", N);
  printf("Hi");
  printf("%i", A[0]);
  hipMalloc((void **) &C,N*sizeof(int));
  multiplyVec<<<1, N>>>(A, B, C);
  hipDeviceSynchronize();
  int sum=0;
  for (int i=0; i<N; i++){
    sum+=C[i];
    printf("Sum %i.  ", sum);
  }
  printf("Computation complete: %i. \n", sum);
  *result=sum;
  return;
}
int main(){
  // take row of A
  // take corresponding column of B
  // take
  printf("A=[");
  for(int i=0; i<sizeof(A)/sizeof(A[0]); i++){
    printf("%i, ", A[i]);
  }
  printf("]\nB=[");
  for(int i=0; i<sizeof(B)/sizeof(B[0]); i++){
   printf("%i, ",B[i]);
  }
  printf("]\nAnswer should be %i.", ans);

  int* cudaResult;
  int result;
  int* cudaA;
  int* cudaB;
  hipMalloc( (void **) &cudaA, 5*sizeof(int));
  hipMalloc( (void **) &cudaB, 5*sizeof(int));
  hipMalloc( (void **) &cudaResult, sizeof(int));
  hipMemcpy(cudaA, (int *) A, 5*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, (int *) B, 5*sizeof(int), hipMemcpyHostToDevice);
  dotproduct<<<1, 1>>>((int *) cudaA, (int *) cudaB, 5, cudaResult);
  hipDeviceSynchronize();
  hipMemcpy(&result, cudaResult, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("%i\n",result);
  return 0;
}
