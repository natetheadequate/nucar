
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void sayHi(){
  printf("Hi\n");
}
__global__ void greet(char* a){
  printf("%s from gpu block %i \n", a, blockIdx.x);
}
#define N 7
int main(){
  printf("Hello World!\n");
  sayHi<<<1,1>>>();
  char* cudaA;
  char greeting[] = "Hello";
  hipMalloc((void **) &cudaA,strlen(greeting)*sizeof(char));
  hipMemcpy(cudaA,greeting,strlen(greeting)*sizeof(char),hipMemcpyHostToDevice);
  greet<<<N,1>>>(cudaA);
  hipFree(cudaA);
  return 0;
}
