
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void sayHi(){
  printf("Hi\n");
}
__global__ void addIntVector(int* a, int* b, int* c){
  printf("hello from gpu block %i", blockIdx.x);
  c[blockIdx.x]=a[blockIdx.x]+b[blockIdx.x];
}
#define N 7
int main(){
  printf("Hello World!\n");
  sayHi<<<N,1>>>();
  //int a[N]= {1,3,5, 9, 11, 13, 15};
  //int b[N]= {0,2,4,6, 8, 10, 12};
  //int c[N]={201,2,3,4,5,6,7};
  int a[N], b[N], c[N];
  for(int i=0; i<N; i++){
    a[i]=2*i;
    b[i]=i;
  }
  int* cudaA;
  int *cudaB;
  int *cudaC;
  hipMalloc((void **) &cudaA,N*sizeof(int));
  hipMalloc((void **) &cudaB,N*sizeof(int));
  hipMalloc((void **) &cudaC,N*sizeof(int));
  hipMemcpy(cudaA,a,N*sizeof(int),hipMemcpyHostToDevice);
  printf("oh boy\n");
  hipMemcpy(cudaB,b,N*sizeof(int),hipMemcpyHostToDevice);
  addIntVector<<<N,1>>>(cudaA,cudaB,cudaC);
  hipDeviceSynchronize();
  printf("whew\n");
  hipMemcpy(c,cudaC,N*sizeof(int),hipMemcpyDeviceToHost);
  printf("%i", hipGetLastError());
  printf("waitwut\n");
  for(int i=0;i<N;i++){
    printf("%i,", c[i]);
  }
  printf("\n");
  hipFree(cudaC);
  hipFree(cudaA);
  hipFree(cudaB);
  return 0;
}
