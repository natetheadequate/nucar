#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>

struct mixednumber{
   int whole;
   int numerator;
   int denominator;
};:ww

__global__ mixednumber divideBase2(int numerator, int denominator){
    struct mixednumber ret = { 
}

__global__ truncadd((void  addends, char* result 

__device__ int modpow(int n, int a, int mod){
  if(a==0){
    return 1;
  }
  return (n*pow(n, a-1)) % mod;
}

__global__ void neobellard(int i, float* result){
   
}
__global__ void bellard(int i, float* result){
  float seq=(pow(2,5) / (-4*i+1))- (1 / (4*i+3))+ (pow(2, 8) / (10 * i +1)) - (pow(2, 6) / (10*i+3)) - (pow(2, 2) / (10*i +5)) - (4/ 10*i+7) + (1 / (10*i+9));
  atomicAdd(result,pow(-1,i)*seq /(pow(2, 10*i)));
  return;
}

int main(int _, char **argv){
  int N=atoi(argv[1]);
  if(N<=0){
    printf("Invalid argument for number of digits to calculate \n");
    return 1;
  }
  if(N>1000){
    printf("I don't want to use that much resources for this. Pick a smaller number");
    return 1;
  }
  printf("Calculating %i digits of pi: \n",N);
  float* cudaResult;
  hipMalloc(&cudaResult, sizeof(float));
  bellard<<<1,N>>>(5,cudaResult);
  hipDeviceSynchronize();
  float result;
  hipMemcpy(&result, cudaResult, sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("%f \n", result*pow(2.0,-6.0));
  return 0;
}
 
