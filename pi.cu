#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>

#define A(n) (4*n+1)
#define B(n) (4*n+3)
#define C(n) (10*n+1)
#define D(n) (10*n+3)
#define E(n) (10*n+5)
#define F(n) (10*n+7)
#define G(n) (10*n+9)

struct fraction {
  unsigned int num;
  unsigned int den;
};

// adds/subtracts atomically the quotient shifted right by bit_offset bits,  as an int, at the given index of res
unsigned int dividerAddSub(unsigned int numerator, unsigned int denominator, unsigned int* res, unsigned int bit_offset, unsigned int index, bool adding){
  unsigned int operand =  ((numerator*sizeof(unsigned int)) >> bit_offset) / denominator;
  if(adding){
  while( (long unsigned int)  (atomicAdd_system(res[index], operand) + operand) > (unsigned int) 1 >> (2,4*sizeof(unsigned int))){
    operand=1;
    index--;
  }
  else while( (double)  (atomicSub_system(res[index], operand) - operand) >  (unsigned int) 1 >>  (2,4*sizeof(unsigned int))){
    operand=1;
    index++;
  }
  return ((numerator*sizeof(unsigned int)) >> bit_offset) % denominator;
 
} 
/*
struct mixednumber{
   int whole;
   int numerator;
   int denominator;


__global__ mixednumber divideBase2(int numerator, int denominator){
    struct mixednumber ret = { 
}

__global__ truncadd((void  addends, char* result 

__device__ int modpow(int n, int a, int mod){
  if(a==0){
    return 1;
  }
  return (n*pow(n, a-1)) % mod;
}

__global__ void neobellard(int i, float* result){
   
}
__global__ void bellard(int i, float* result){
  float seq=(pow(2,5) / (-4*i+1))- (1 / (4*i+3))+ (pow(2, 8) / (10 * i +1)) - (pow(2, 6) / (10*i+3)) - (pow(2, 2) / (10*i +5)) - (4/ 10*i+7) + (1 / (10*i+9));
  atomicAdd(result,pow(-1,i)*seq /(pow(2, 10*i)));
  return;
}

int main(int _, char **argv){
  int N=atoi(argv[1]);
  if(N<=0){
    printf("Invalid argument for number of digits to calculate \n");
    return 1;
  }
  if(N>1000){
    printf("I don't want to use that much resources for this. Pick a smaller number");
    return 1;
  }
  printf("Calculating %i digits of pi: \n",N);
  float* cudaResult;
  hipMalloc(&cudaResult, sizeof(float));
  bellard<<<1,N>>>(5,cudaResult);
  hipDeviceSynchronize();
  float result;
  hipMemcpy(&result, cudaResult, sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("%f \n", result*pow(2.0,-6.0));
  return 0;
}
*/

int main(){
  int res[50];
  assert(dividerAddSub((unsigned int) 1,(unsigned int) 2, res,(unsigned int)0,(unsigned int)2, true)==0);
  assert(res[2] == (unsigned int) 1 << (sizeof(unsigned int)*4-1));
  assert(dividerAddSub((unsigned int) 1,(unsigned int) 3, res,(unsigned int)2,(unsigned int)3, true)==1);
  assert(res[3] == (unsigned int)1 << (sizeof(unsigned int)*4-1));
  return 0;
}

}
 
